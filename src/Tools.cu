#include "hip/hip_runtime.h"
#include "Tools.h"

namespace BioAlign{
    ////////////////////
    // Tool functions //
    ////////////////////

    void LowerNode(Node *nd_ptr){
        char *seq_device, *seq_host;
        int sz = nd_ptr->Len() * sizeof(char);

        int thread_num = 512, block_num = (sz / thread_num) + 1;

        seq_host = (char*)malloc(sz + 1);

        hipMalloc(&seq_device, sz);
        hipMemcpy(seq_device, nd_ptr->Sequence(), sz, hipMemcpyHostToDevice);

        Lowercase<<<block_num, thread_num>>>(seq_device, sz);

        hipMemcpy(seq_host, seq_device, sz, hipMemcpyDeviceToHost);

        nd_ptr->Update(seq_host);
    }

    void UpperNode(Node *nd_ptr){
        char *seq_device, *seq_host;
        int sz = nd_ptr->Len() * sizeof(char);

        int thread_num = 512, block_num = (sz / thread_num) + 1;

        seq_host = (char*)malloc(sz + 1);

        hipMalloc(&seq_device, sz);
        hipMemcpy(seq_device, nd_ptr->Sequence(), sz, hipMemcpyHostToDevice);

        Uppercase<<<block_num, thread_num>>>(seq_device, sz);

        hipMemcpy(seq_host, seq_device, sz, hipMemcpyDeviceToHost);

        nd_ptr->Update(seq_host);

        free(seq_host);
        hipFree(seq_device);
    }

    void ReadFasta(std::string fname, std::vector<Node*>& nodes_ptr){
        try{
            std::size_t pos = fname.find_last_of('.');
            std::string file_extension = fname.substr(pos + 1);

            if((file_extension.compare("fa") != 0) && (file_extension.compare("fasta") != 0))
                throw "In ReadFasta(std::string, std::vector<Node*>&) : File type not supported.";

            std::ifstream input_file;
            std::string line;

            input_file.open(fname);
            if(!input_file)
                throw "In ReadFasta(std::string, std::vector<Node*>&) : Cannot open file. Maybe wrong file name is given.";

            while(!input_file.eof()){
                std::getline(input_file, line);
                if(line[0] == '>')
                    std::getline(input_file, line);

                nodes_ptr.push_back(new Node(line));
            }

            input_file.close();
        }
        catch(const char* error_message){
            std::cout << error_message << std::endl;
        }
    }

    double FindDirectedDistance(Node *ndptr_a, Node *ndptr_b){
        int n = ndptr_a->Len(), m = ndptr_b->Len(), sum;
        int *lens_ab, *lens_aa, *lens_device;
        char *str_a_device, *str_b_device;
        double result, l_ab, l_aa;

        int thread_num = 512, block_num = (n / thread_num) + 1;

        lens_ab = (int*)malloc(n * sizeof(int)); // lens_ab[a(0), a(1), ...]. a <-> A. sum(lens_ab) / |A|.
        lens_aa = (int*)malloc(n * sizeof(int));

        hipMalloc(&lens_device, n * sizeof(int));
        hipMalloc(&str_a_device, n * sizeof(char));
        hipMalloc(&str_b_device, m * sizeof(char));

        hipMemcpy(str_a_device, ndptr_a->Sequence(), n * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(str_b_device, ndptr_b->Sequence(), m * sizeof(char), hipMemcpyHostToDevice);

        LongestSubstringLen<<<block_num, thread_num>>>(lens_device, str_a_device, str_b_device, n, m);
        hipMemcpy(lens_ab, lens_device, n * sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i < n; i ++)
            lens_aa[n - i - 1] = i + 1;

        sum = 0;
        for(int i = 0; i < n; i ++)
            sum += lens_ab[i];
        l_ab = (double)sum / n; // L(A, B). Larger L(A, B) -> Similar A, B.

        sum = 0;
        for(int i = 0; i < n; i ++)
            sum += lens_aa[i];
        l_aa = (double)sum / n; // L(A, A)

        result = (log(m) / l_ab) - (log(n) / l_aa); // result = d(A, B).

        free(lens_ab);
        free(lens_aa);
        hipFree(lens_device);
        hipFree(str_a_device);
        hipFree(str_b_device);

        return result;
    }
};