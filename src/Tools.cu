#include "Tools.h"

namespace BioAlign{
    ////////////////////
    // Tool functions //
    ////////////////////

    void LowerNode(Node *nd_ptr){
        char *seq_device, *seq_host;
        int sz = nd_ptr->Len() * sizeof(char);

        int thread_num = 512, block_num = (sz / thread_num) + 1;

        seq_host = (char*)malloc(sz + 1);

        hipMalloc(&seq_device, sz);
        hipMemcpy(seq_device, nd_ptr->Sequence(), sz, hipMemcpyHostToDevice);

        Lowercase<<<block_num, thread_num>>>(seq_device, sz);

        hipMemcpy(seq_host, seq_device, sz, hipMemcpyDeviceToHost);

        nd_ptr->Update(seq_host);
    }

    void UpperNode(Node *nd_ptr){
        char *seq_device, *seq_host;
        int sz = nd_ptr->Len() * sizeof(char);

        int thread_num = 512, block_num = (sz / thread_num) + 1;

        seq_host = (char*)malloc(sz + 1);

        hipMalloc(&seq_device, sz);
        hipMemcpy(seq_device, nd_ptr->Sequence(), sz, hipMemcpyHostToDevice);

        Uppercase<<<block_num, thread_num>>>(seq_device, sz);

        hipMemcpy(seq_host, seq_device, sz, hipMemcpyDeviceToHost);

        nd_ptr->Update(seq_host);
    }
};